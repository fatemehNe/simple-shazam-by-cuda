#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <iomanip>
#include<vector>
#include <fstream>
#include <windows.h>
#include <omp.h>

using namespace std;

#define cudaErrchk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void comparator(string songDir, string sampleDir);
double* cudaSizeCalc(hipfftComplex* array, int size);
double cudaCosSim(double* song, double* sample, int size);
vector<string> get_file_names_in_folder(string folder);


__global__ void complexSizeKernel(hipfftComplex *cmplx, double *dblArray, int size) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size)
		dblArray[i] = sqrt(pow(cmplx[i].x, 2) + pow(cmplx[i].y, 2));
}

__global__ void normalization(double *dblArray, double *newdblArray, double* minMax, int size) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size)
		newdblArray[i] = (dblArray[i] - minMax[0]) / (minMax[1] - minMax[0]);
}

__global__ void maxByReduction(double *array, double *max, int size) {
	extern __shared__ double maxData[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size)
		maxData[tid] = array[i];
	else
		maxData[tid] = 0;

	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s) {
			if (maxData[tid] < maxData[tid + s])
				maxData[tid] = maxData[tid + s];
		}
		__syncthreads();
	}
	// write max for this block to global mem
	if (tid == 0) {
		max[blockIdx.x] = maxData[0];
	}
}

__global__ void minByReduction(double *array, double *min, int size) {
	extern __shared__ double minData[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size)
		minData[tid] = array[i];
	else
		minData[tid] = 10000;


	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s) {
			if (minData[tid] > minData[tid + s])
				minData[tid] = minData[tid + s];
		}
		__syncthreads();
	}
	// write min for this block to global mem
	if (tid == 0) {
		min[blockIdx.x] = minData[0];
	}
}

__global__ void multiplyReduction(double *song, double *sample, double *result, int size) {
	extern __shared__ double multiply[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size)
		multiply[tid] = song[i] * sample[i];
	else
		multiply[tid] = 0;

	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s) {
			multiply[tid] += multiply[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) {
		result[blockIdx.x] = multiply[0];
	}
}

__global__ void vectorSize(double *song, double *result, int arraySize) {
	extern __shared__ double size[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < arraySize)
		size[tid] = pow(song[i], 2);
	else
		size[tid] = 0;

	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1)
	{
		if (tid < s) {
			size[tid] += size[tid + s];
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) {
		result[blockIdx.x] = size[0];
	}
}

double cosineSimilarity(double* a, double* b, int sampleSize) {
	double cosmul = 0.0;
	double asize = 0.0;
	double bsize = 0.0;
	for (int i = 0; i < sampleSize; i++) {
		cosmul += a[i] * b[i];
		asize += pow(a[i], 2);
		bsize += pow(b[i], 2);
	}
	cosmul = abs(cosmul);
	asize = sqrt(asize);
	bsize = sqrt(bsize);

	return cosmul / (asize * bsize);
}

double calculateSize(hipfftComplex in) {
	return sqrt(pow(in.x, 2) + pow(in.y, 2));
}


double* normalize(hipfftComplex* array, int size) {
	double min = 100000.0;
	double max = -100000.0;
	double* newArray = (double *)malloc(sizeof(double)*size);
	int i = 0;
	for (i = 0; i < size; i++) {
		newArray[i] = calculateSize(array[i]);
		if (min > newArray[i]) {
			min = newArray[i];
		}
		if (max < newArray[i]) {
			max = newArray[i];
		}
	}

	for (i = 0; i < size; i++) {
		newArray[i] = (newArray[i] - min) / (max - min);
	}

	return newArray;
}

hipfftComplex* FourierCalc(hipfftComplex* cmplxArray, int size) {
	hipfftHandle plan;
	hipfftComplex *data;

	cudaErrchk(hipMalloc((void**)&data, sizeof(hipfftComplex)*size));

	cudaErrchk(hipMemcpy(data, cmplxArray, sizeof(hipfftComplex)*size, hipMemcpyHostToDevice));

	if (hipfftPlan1d(&plan, size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		exit(1);
	}

	if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		exit(1);
	}


	cudaErrchk(hipDeviceSynchronize());

	cudaErrchk(hipMemcpy(cmplxArray, data, sizeof(hipfftComplex)*size, hipMemcpyDeviceToHost));

	hipfftDestroy(plan);
	hipFree(data);
	return cmplxArray;
}


hipfftComplex* readData(const char* path, int &size) {

	ifstream inFile;
	float fdata;

	inFile.open(path);
	if (!inFile) {
		cout << "Unable to open file";
		exit(1);
	}

	size = 0;
	while (inFile >> fdata) {
		size++;
	}
	inFile.close();
	inFile.open(path);
	if (!inFile) {
		cout << "Unable to open file";
		exit(1);
	}

	hipfftComplex *audioData = (hipfftComplex *)malloc(sizeof(hipfftComplex)*size);

	int i = 0;
	while (inFile >> fdata && i < size) {

		audioData[i].x = fdata;
		audioData[i].y = 0;
		i++;
	}

	inFile.close();
	return audioData;
}

int main(int argc, char* argv[]) {

	string Songdirectory = argv[1];//"C:\\Users\\ahmad\\Desktop\\cudame\\final\\songs\\";
	string sampledirectory = argv[2];//"C:\\Users\\ahmad\\Desktop\\cudame\\final\\songs\\6080.txt";
	comparator(Songdirectory, sampledirectory);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void comparator(string songDir, string sampleDir)
{
	vector<string> sampleNames = get_file_names_in_folder(sampleDir);
	for (int z = 0; z < sampleNames.size(); z++)
	{
		int songSize = 0;
		int sampleSize = 0;
		string similarSong;
		double mainMax = 0;
		hipfftComplex * sampleData = readData((sampleDir + sampleNames.at(z)).c_str(), sampleSize);

		hipfftComplex * sampleFourier = FourierCalc(sampleData, sampleSize);

		double* sampleSizeData = cudaSizeCalc(sampleFourier, sampleSize);

		int index = 100000;// (int)sampleSize / 4000;

		hipfftComplex *songSection = (hipfftComplex *)malloc(sizeof(hipfftComplex)*sampleSize);

		vector<string> textNames = get_file_names_in_folder(songDir);

		for (int i = 0; i < textNames.size(); i++)
		{
			//printf("%s: \n", textNames.at(i));
			hipfftComplex * song = readData((songDir + textNames.at(i)).c_str(), songSize);
			double max = 0;

			int x = 0;

			while (x < songSize) {
				if (songSize  <  sampleSize + x)
					break;


#pragma omp parallel for 
				for (int j = x; j < x + sampleSize; j++) {
					songSection[j - x] = song[j];
				}

				songSection = FourierCalc(songSection, sampleSize);
				double* songSizeSection = cudaSizeCalc(songSection, sampleSize);
				double val = cudaCosSim(songSizeSection, sampleSizeData, sampleSize);
				if (max < val) {
					max = val;
				}
				x += index;
			}
			// printf("song name: %s , sample name : %s %f\n", textNames.at(i), sampleNames.at(z), max);
			if (mainMax < max) {
				mainMax = max;
				similarSong = textNames.at(i);
			}
		}
		if (mainMax > 0.5)
			printf("similar song name: %s , sample name : %s %f\n", similarSong, sampleNames.at(z), mainMax);
		else
			printf("for  %s no song found\n", sampleNames.at(z));
	}
}

vector<string> get_file_names_in_folder(string folder) {
	vector<string> names;
	string search_path = folder + "/*.*";
	WIN32_FIND_DATA fd;
	HANDLE hFind = ::FindFirstFile(search_path.c_str(), &fd);
	if (hFind != INVALID_HANDLE_VALUE) {
		do {
			// read all (real) files in current folder
			// , delete '!' read other 2 default folder . and ..
			if (!(fd.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY)) {
				names.push_back(fd.cFileName);
			}
		} while (::FindNextFile(hFind, &fd));
		::FindClose(hFind);
	}
	return names;
}

double* cudaSizeCalc(hipfftComplex* cmplxarray, int size) {
	int threadNum = 128;
	int blockNum = ceil(size / threadNum);

	hipfftComplex *dev_CmplxArray;
	double *dev_dblArray;

	double* host_newDbl = (double *)malloc(sizeof(double)*  size);


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaErrchk(hipSetDevice(0));

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaErrchk(hipMalloc((void**)&dev_dblArray, size * sizeof(double)));

	cudaErrchk(hipMalloc((void**)&dev_CmplxArray, size * sizeof(hipfftComplex)));

	// Copy input vectors from host memory to GPU buffers.
	cudaErrchk(hipMemcpy(dev_CmplxArray, cmplxarray, size * sizeof(hipfftComplex), hipMemcpyHostToDevice));

	//block numbers , thread bnumbers 

	complexSizeKernel << <blockNum, threadNum >> > (dev_CmplxArray, dev_dblArray, size);

	// Check for any errors launching the kernel
	cudaErrchk(hipGetLastError());

	cudaErrchk(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	cudaErrchk(hipMemcpy(host_newDbl, dev_dblArray, size * sizeof(double), hipMemcpyDeviceToHost));

	hipFree(dev_CmplxArray);
	hipFree(dev_dblArray);

	return host_newDbl;
}

double cudaCosSim(double* song, double* sample, int size) {

	int threadNum = 128;
	int blockNum = ceil(size / threadNum);

	const int streamsNum = 2;
	hipStream_t streams[streamsNum];
	hipStreamCreate(&streams[0]);
	hipStreamCreate(&streams[1]);

	double *dev_song;
	double *dev_sample;
	double *dev_result;
	double *dev_sng;
	double *dev_smp;

	double multiplication = 0.0;
	double size1 = 0.0;
	double size2 = 0.0;
	double similarity;

	double* host_result = (double *)malloc(sizeof(double)*  blockNum);
	double* host_sng = (double *)malloc(sizeof(double)*  blockNum);
	double* host_smp = (double *)malloc(sizeof(double)*  blockNum);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaErrchk(hipSetDevice(0));
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaErrchk(hipMalloc((void**)&dev_song, size * sizeof(double)));

	cudaErrchk(hipMalloc((void**)&dev_sample, size * sizeof(double)));

	cudaErrchk(hipMalloc((void**)&dev_result, blockNum * sizeof(double)));

	cudaErrchk(hipMalloc((void**)&dev_sng, blockNum * sizeof(double)));

	cudaErrchk(hipMalloc((void**)&dev_smp, blockNum * sizeof(double)));

	// Copy input vectors from host memory to GPU buffers.
	cudaErrchk(hipMemcpy(dev_sample, sample, size * sizeof(double), hipMemcpyHostToDevice));

	// Copy input vectors from host memory to GPU buffers.
	cudaErrchk(hipMemcpy(dev_song, song, size * sizeof(double), hipMemcpyHostToDevice));

	//block numbers , thread bnumbers 
	size_t shm_size = threadNum * sizeof(double);
	multiplyReduction << <blockNum, threadNum, shm_size >> > (dev_song, dev_sample, dev_result, size);

	// Check for any errors launching the kernel
	cudaErrchk(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaErrchk(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	cudaErrchk(hipMemcpy(host_result, dev_result, blockNum * sizeof(double), hipMemcpyDeviceToHost));

	cudaErrchk(hipGetLastError());

	vectorSize << <blockNum, threadNum, shm_size, streams[0] >> > (dev_sample, dev_smp, size);

	// Check for any errors launching the kernel
	cudaErrchk(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaErrchk(hipDeviceSynchronize());

	cudaErrchk(hipMemcpy(host_smp, dev_smp, blockNum * sizeof(double), hipMemcpyDeviceToHost));

	cudaErrchk(hipGetLastError());

	vectorSize << <blockNum, threadNum, shm_size, streams[1] >> > (dev_song, dev_sng, size);

	// Check for any errors launching the kernel
	cudaErrchk(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaErrchk(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	cudaErrchk(hipMemcpy(host_sng, dev_sng, blockNum * sizeof(double), hipMemcpyDeviceToHost));

	// Check for any errors launching the kernel
	cudaErrchk(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaErrchk(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.

#pragma omp parallel for reduction( +: multiplication, size1 , size2 )
	for (int i = 0; i < blockNum; i++) {
		multiplication += host_result[i];
		size1 += host_sng[i];
		size2 += host_smp[i];
	}

	multiplication = abs(multiplication);
	size1 = sqrt(size1);
	size2 = sqrt(size2);

	similarity = multiplication / (size1 * size2);

	hipFree(dev_song);
	hipFree(dev_sample);
	hipFree(dev_result);
	hipFree(dev_sng);
	hipFree(dev_smp);

	return similarity;
}
